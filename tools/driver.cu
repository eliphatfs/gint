#include <hip/hip_runtime.h>
#include <stdio.h>


#define CHECK_CUDA_DRIVER(call)                                               \
do {                                                                          \
    hipError_t result = call;                                                   \
    if (result != hipSuccess) {                                             \
        const char* errorStr = NULL;                                          \
        const char* errorName = NULL;                                         \
        hipDrvGetErrorName(result, &errorName);                                   \
        hipDrvGetErrorString(result, &errorStr);                                  \
        fprintf(stderr, "CUDA Driver API Error at %s:%d - %s (%s)\n",         \
                __FILE__, __LINE__,                                           \
                errorName ? errorName : "Unknown",                            \
                errorStr ? errorStr : "Unknown CUDA error");                  \
        exit(EXIT_FAILURE); /* Or handle error appropriately */               \
    }                                                                         \
} while(0)


void * read_ptx(const char* filename)
{
    void * buffer = 0;
    long length;
    FILE * f = fopen (filename, "rb");

    if (f)
    {
        fseek (f, 0, SEEK_END);
        length = ftell (f);
        fseek (f, 0, SEEK_SET);
        buffer = calloc (length + 1, sizeof(char));
        if (buffer)
        {
            fread (buffer, 1, length, f);
        }
        fclose (f);
    }
    return buffer;
}


int main()
{
    hipDevice_t device;
    hipCtx_t context;

    // Initialize the driver API
    CHECK_CUDA_DRIVER(hipInit(0));
    // Get a handle to the first compute device
    CHECK_CUDA_DRIVER(hipDeviceGet(&device, 0));
    // Create a compute device context
    CHECK_CUDA_DRIVER(hipCtxCreate(&context, 0, device));

    hipModule_t module;
    hipFunction_t function;
    void * ptx = read_ptx("test.ptx");

    // JIT compile a null-terminated PTX string
    CHECK_CUDA_DRIVER(hipModuleLoadData(&module, ptx));

    // Get a handle to the "myfunction" kernel function
    CHECK_CUDA_DRIVER(hipModuleGetFunction(&function, module, "geval"));

    int x = 0;

    void* kernelParams[1] = {&x};

    CHECK_CUDA_DRIVER(hipModuleLaunchKernel(
        function,
        1, 1, 1,
        1, 1, 1,
        0,
        (hipStream_t)0,
        kernelParams,
        NULL
    ));
    CHECK_CUDA_DRIVER(hipStreamSynchronize((hipStream_t)0));

    free(ptx);
    
    CHECK_CUDA_DRIVER(hipModuleUnload(module));
    CHECK_CUDA_DRIVER(hipCtxDestroy(context));
    return 0;
}
